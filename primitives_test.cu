#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "hip/device_functions.h"
#include <thrust/scan.h>
#include "CycleTimer.h"
#define SCAN_BLOCK_DIM 512 
#define uint unsigned int
#include "exclusiveScan.cu_inl"
#include "cuPrintf.cu"
#include "scan.cu"

using namespace std;
extern float toBW(int bytes, float sec);

__global__ void prescan(int *g_odata, int *g_idata, int n){

    extern __shared__ int temp[];// allocated on invocation
    int thid = threadIdx.x;
    int offset = 1;
    temp[2*thid] = g_idata[2*thid]; // load input into shared memory
    temp[2*thid+1] = g_idata[2*thid+1];
    for (int d = n>>1; d > 0; d >>= 1){ // build sum in place up the tree
        __syncthreads();
        if (thid < d){

            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    if (thid == 0) { temp[n - 1] = 0; } // clear the last element
    for (int d = 1; d < n; d *= 2){ // traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (thid < d){
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    g_odata[2*thid] = temp[2*thid]; // write results to device memory
    g_odata[2*thid+1] = temp[2*thid+1];
}

__global__ void
primitive_select_kernel(int N, int* tuples, int* result, int* result_size) {

	__shared__ uint input[SCAN_BLOCK_DIM];
	__shared__ uint output[SCAN_BLOCK_DIM];
	__shared__ uint scratch[2 * SCAN_BLOCK_DIM];

	int threadIndex =  threadIdx.x;
	int partition = blockIdx.x *  blockDim.x;
	//cuPrintf("%d\n", threadIndex);
	input[threadIndex] = 0;
	output[threadIndex] = 0;
 	if ( partition + threadIndex < N ) {
		input[threadIndex] = tuples[partition + threadIndex] % 2 == 0? 1 : 0;
 	}
	 __syncthreads();
	 sharedMemExclusiveScan(threadIndex, input, output, scratch, SCAN_BLOCK_DIM);
	if(input[threadIndex]){
		 //atomicAdd(result_size + blockIdx.x, 1);
    	 result[partition + output[threadIndex]] = tuples[partition + threadIndex];
 	}
     //__syncthreads();
    // reduction phase
    extern __shared__ int sdata;
    unsigned int i = partition + threadIndex;
    int x = 0;
    if(i < n)
      {
        x = input[i];
      }
    sdata[threadIdx.x] = x;
    __syncthreads();
    for(int offset = blockDim.x / 2;
      offset > 0;
      offset >>= 1)
  {
    if(threadIdx.x < offset)
    {
      // add a partial sum upstream to our own
      sdata[threadIdx.x] += sdata[threadIdx.x + offset];
    }

    // wait until all threads in the block have
    // updated their partial sums
    __syncthreads();
  }

  // thread 0 writes the final result
  if(threadIdx.x == 0)
  {
    result_size[blockIdx.x] = sdata[0];
  }
}

__global__ void coalesced(int N, int* result, int* result_size, int* histogram, int* out) {
	int threadIndex =  threadIdx.x;
	int partition = blockIdx.x *  blockDim.x;
    if( threadIndex < result_size[blockIdx.x] ) {
		out[histogram[blockIdx.x] + threadIndex] = result[partition + threadIndex];
	}
	__syncthreads();
}

void
primitive_scan(int N, int inData[], int outData[]) {
	int large_num = 512;
    float tmp[large_num];
    float* large_in;
    float* large_out;

	hipMalloc((void**) &large_in, sizeof(float) * large_num);
	hipMalloc((void**) &large_out, sizeof(float) * large_num);
    
    for(int i = 0; i < large_num; i ++) {
        tmp[i] = 1.0;
    }
	hipMemcpy(large_in, tmp, sizeof(float) * large_num, hipMemcpyHostToDevice);
    preallocBlockSums(large_num);
    prescanArray(large_out, large_in, large_num);
	hipMemcpy(tmp, large_out, sizeof(float) * large_num, hipMemcpyDeviceToHost);
    for(int i = 0; i < large_num; i ++) {
        printf("%f ", tmp[i]);
    }
    printf("\n");
}

void 
primitive_select(int N, int inData[], int outData[]) {
	const int threadPerBlock = 512;
	const int blocks = (N + threadPerBlock - 1) / threadPerBlock;
	const int blocksOfReulstSize = ( blocks + threadPerBlock - 1) / threadPerBlock;
    int totalBytes = N * sizeof(int) * 2;
    printf("Num of tuples %d\n", N);
	printf("Num of blocks %d\n", blocks);
	printf("Num of blocks for result size %d\n", blocksOfReulstSize);
    
    int* device_in;
	int* device_result;
	int* result_size;
	int* histogram;
	int* out;
	int* tmp = (int*)calloc(N, sizeof(int));
    double startTime = CycleTimer::currentSeconds();
	hipMalloc((void**) &device_in, sizeof(int) * N);
	hipMalloc((void**) &device_result, sizeof(int) * N);
	hipMalloc((void**) &out, sizeof(int) * N);
	hipMalloc((void**) &result_size, sizeof(int) * blocks);
	hipMalloc((void**) &histogram, sizeof(int) * blocks);

	hipMemcpy(device_in, inData, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(device_result, tmp, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(out, tmp, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(result_size, tmp, sizeof(int) * blocks, hipMemcpyHostToDevice);
    cudaPrintfInit();
    double startTime_inner = CycleTimer::currentSeconds();	
	primitive_select_kernel<<<blocks, threadPerBlock>>>(N, device_in, device_result, result_size);

   // int test_result_size[blocks];
   // hipMemcpy(test_result_size, result_size, sizeof(int)*blocks, hipMemcpyDeviceToHost);
   // for(int i = 0 ; i < blocks ; i ++) {
   //     printf("%d, ", test_result_size[i]);
   // }
   // printf("\n");
	hipDeviceSynchronize();
	//prescan<<<blocksOfReulstSize, threadPerBlock, blocks * threadPerBlock * 2 * sizeof(int)>>>(histogram, result_size, blocks);
    
    thrust::device_ptr<int> dev_ptr1(result_size);
    thrust::device_ptr<int> dev_ptr2(histogram);
    thrust::exclusive_scan(dev_ptr1, dev_ptr1 + blocks, dev_ptr2);
   // int test_histgram[blocks];
   // hipMemcpy(test_histgram, histogram, sizeof(int)*blocks, hipMemcpyDeviceToHost);
   // for(int i = 0 ; i < blocks; i ++) {
   //     printf("%d, ", test_histgram[i]);
   // }
   // printf("\n");
	coalesced<<<blocks, threadPerBlock>>>(N, device_result, result_size, histogram, out);
    double endTime_inner = CycleTimer::currentSeconds();
    cudaPrintfDisplay(stdout, true);
 	cudaPrintfEnd();
    hipMemcpy(outData, out, sizeof(int) * N, hipMemcpyDeviceToHost);
    double endTime = CycleTimer::currentSeconds();
    
    double overallDuration = endTime - startTime;
    double kernelDuration = endTime_inner - startTime_inner;
    
    printf("CUDA overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("CUDA execution time for kernel: %.3f ms\t\t[%.3f GB/s]\n", 1000.f*kernelDuration, toBW(totalBytes, kernelDuration));
    hipFree(device_in);
    hipFree(device_result);
    hipFree(out);
    hipFree(result_size);
    hipFree(histogram);
}	

void
printCudaInfo() {
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
}
