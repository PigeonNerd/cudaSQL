#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "hip/device_functions.h"
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "CycleTimer.h"
#define SCAN_BLOCK_DIM 512
#define uint unsigned int
#include "exclusiveScan.cu_inl"
#include "cuPrintf.cu"
#include "scan.cu"
#include "book.h"

#define GRID_DIM 65535

using namespace std;
extern float toBW(int bytes, float sec);


//brute force find matching tuples
void sequential_join(int2* rel_a, int2* rel_b, int rel_a_size, int rel_b_size, int3* out, int* numResult) {
   *numResult = 0;
   double startTime = CycleTimer::currentSeconds();
     for (int i = 0; i < rel_a_size; i++) {
         for(int j = 0; j < rel_b_size; j ++ ) {
            if(rel_a[i].x == rel_b[j].x) {
                out[*numResult].x = rel_a[i].x;
                out[*numResult].y = rel_a[i].y;
                out[*numResult].z = rel_b[j].y;
                (*numResult)++;
            }
            else if( rel_a[i].x < rel_b[j].x) {
                break;
            }
        }
     }
   double endTime = CycleTimer::currentSeconds();
   printf("time excution from sequential join %.3f ms\n",1000.f * (endTime  - startTime));
   printf("sequential join produces %d tuples\n", *numResult);
}

void validate_join(int3* seq_out, int seq_num, int3* cuda_out, int cuda_num) {
    if( seq_num != cuda_num ) {
        printf("num of tuples seq(%d) != cuda(%d)", seq_num, cuda_num);
        return;
    }
    for(int i = 0 ; i < cuda_num; i ++) {
       if( seq_out[i].x != cuda_out[i].x ||seq_out[i].y != cuda_out[i].y || seq_out[i].z != cuda_out[i].z) {
            printf("At line %d, not match FAIL\n",i);
            printf("seq: [%d, %d, %d], GPU: [%d, %d, %d]\n",seq_out[i].x,seq_out[i].y, seq_out[i].z, cuda_out[i].x, cuda_out[i].y, cuda_out[i].z);
            return;
       }
    }
    printf("JOIN PASS !\n");
}

/*
    choose the quilified tuples from the relation
    and get the cout of tuples of each block
*/
__global__ void
primitive_select_kernel(int N, int blocks, int* tuples, int* result, int* result_size) {

	__shared__ uint input[SCAN_BLOCK_DIM];
	__shared__ uint output[SCAN_BLOCK_DIM];
	__shared__ uint scratch[2 * SCAN_BLOCK_DIM];
	int threadIndex =  threadIdx.x;
	int partition = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    int blockIndex = blockIdx.y * gridDim.x + blockIdx.x;

    if (partition + threadIndex >= N || blockIndex >= blocks) {
        return;
    }

	//cuPrintf("%d %d %d\n", blockIdx.y, gridDim.x, blockIndex);
	input[threadIndex] = 0;
	output[threadIndex] = 0;
 	if ( partition + threadIndex < N ) {
		input[threadIndex] = tuples[partition + threadIndex] % 2 == 0? 1 : 0;
 	}
	 __syncthreads();
	 sharedMemExclusiveScan(threadIndex, input, output, scratch, SCAN_BLOCK_DIM);
	if(input[threadIndex]){
		 //atomicAdd(result_size + blockIndex, 1);
    	 result[partition + output[threadIndex]] = tuples[partition + threadIndex];
 	}
    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset) {
          // add a partial sum upstream to our own
          input[threadIdx.x] += input[threadIdx.x + offset];
     }

        // wait until all threads in the block have
        // updated their partial sums
        __syncthreads();
      }
      // thread 0 writes the final result
      if(threadIdx.x == 0) {
          result_size[blockIndex] = input[0];
      }
}

/*
    gather stage
*/
__global__ void coalesced(int N, int* result, int* result_size, int* histogram, int* out) {
	int threadIndex =  threadIdx.x;
	int partition = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    int blockIndex = blockIdx.y * gridDim.x + blockIdx.x;

    if (partition + threadIndex >= N) {
        return;
    }

    if( threadIndex < result_size[blockIndex]) {
          /*if(histogram[blockIndex] + threadIndex == 0) {
            cuPrintf("%d %d\n", blockIndex, threadIndex);
          }*/
		  out[histogram[blockIndex] + threadIndex] = result[partition + threadIndex];
	   }
	__syncthreads();
}


/*
    This is a sample of how to use scanLargeArray
    from Nvidia SDK
*/
void primitive_scan(int N, int inData[], int outData[]) {
	int large_num = 39063;
    float tmp[large_num];
    float* large_in;
    float* large_out;
    double startTime;
    double endTime;
	hipMalloc((void**) &large_in, sizeof(float) * large_num);
	hipMalloc((void**) &large_out, sizeof(float) * large_num);
    //hipMemset(large_in, 1, large_num * sizeof(float));
    for(int i = 0; i < large_num; i ++) {
        tmp[i] = 1.0;
    }
	hipMemcpy(large_in, tmp, sizeof(float) * large_num, hipMemcpyHostToDevice);
    startTime = CycleTimer::currentSeconds();
    preallocBlockSums(large_num);
    prescanArray(large_out, large_in, large_num);
    endTime = CycleTimer::currentSeconds();
   printf("time excution from large array scan %.3f ms\n", 1000.f * (endTime  - startTime));
   /* startTime = CycleTimer::currentSeconds();
    thrust::device_ptr<float> dev_ptr1(large_in);
    thrust::device_ptr<float> dev_ptr2(large_out);
    thrust::exclusive_scan(dev_ptr1, dev_ptr1 + large_num, dev_ptr2);
    endTime = CycleTimer::currentSeconds();
   printf("time excution from thrust scan %.3f ms\n",1000.f * (endTime  - startTime));*/
    hipMemcpy(tmp, large_out, sizeof(float) * large_num, hipMemcpyDeviceToHost);
    for(int i = 0; i < large_num; i ++) {
        printf("%f ", tmp[i]);
    }
    printf("\n");
    int y[] = {1, 2};
    printf("%d\n", y[(int)tmp[1]]);
    deallocBlockSums();
}


/*
    Implementation of SELECT operation
*/
void
primitive_select(int N, int inData[], int outData[]) {
	const int threadPerBlock = 512;
	const int blocks = (N + threadPerBlock - 1) / threadPerBlock;

  int rows = (blocks / GRID_DIM) == 0? 1 : (blocks / GRID_DIM) + 1;
  int cols = (blocks / GRID_DIM) == 0? blocks : GRID_DIM;
  dim3 blockDim(threadPerBlock, 1);
  dim3 gridDim(cols, rows);
    printf("rows %d cols: %d\n", rows, cols);
	const int blocksOfReulstSize = ( blocks + threadPerBlock - 1) / threadPerBlock;
    int totalBytes = N * sizeof(int) * 2;
    printf("Num of tuples %d\n", N);
	printf("Num of blocks %d\n", blocks);
	printf("Num of blocks for result size %d\n", blocksOfReulstSize);
    int* device_in;
	int* device_result;
	int* result_size;
	int* histogram;
	int* out;
	int* tmp = (int*)calloc(N, sizeof(int));
	hipMalloc((void**) &device_in, sizeof(int) * N);
	hipMalloc((void**) &device_result, sizeof(int) * N);
	hipMalloc((void**) &out, sizeof(int) * N);
	hipMalloc((void**) &result_size, sizeof(int) * blocks);
	hipMalloc((void**) &histogram, sizeof(int) * blocks);
    double startTime = CycleTimer::currentSeconds();
	hipMemcpy(device_in, inData, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(device_result, tmp, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(out, tmp, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(result_size, tmp, sizeof(int) * blocks, hipMemcpyHostToDevice);
    cudaPrintfInit();
    double startTime_inner = CycleTimer::currentSeconds();
//	for(int i = 0 ; i < 10 ; i ++) {
    primitive_select_kernel<<<gridDim, blockDim>>>(N, blocks, device_in, device_result, result_size);

   /* int test_result_size[blocks];
    hipMemcpy(test_result_size, result_size, sizeof(int) * blocks, hipMemcpyDeviceToHost);
    for(int i = 0 ; i < blocks ; i ++) {
       printf("%d: %d, ",i ,test_result_size[i]);
    }
    printf("\n");
	hipDeviceSynchronize();*/

    thrust::device_ptr<int> dev_ptr1(result_size);
    thrust::device_ptr<int> dev_ptr2(histogram);
    thrust::exclusive_scan(dev_ptr1, dev_ptr1 + blocks, dev_ptr2);
   /* int test_histgram[blocks];
    hipMemcpy(test_histgram, histogram, sizeof(int)*blocks, hipMemcpyDeviceToHost);
    for(int i = 0 ; i < blocks; i ++) {
        printf("%d, ", test_histgram[i]);
    }
    printf("\n");*/
	coalesced<<<gridDim, blockDim>>>(N, device_result, result_size, histogram, out);
  //  }
    double endTime_inner = CycleTimer::currentSeconds();
    cudaPrintfDisplay(stdout, true);
 	  cudaPrintfEnd();

    hipMemcpy(outData, out, sizeof(int) * N, hipMemcpyDeviceToHost);
    double endTime = CycleTimer::currentSeconds();

    double overallDuration = endTime - startTime;
    double kernelDuration = endTime_inner - startTime_inner;
    printf("CUDA overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("CUDA execution time for kernel: %.3f ms\t\t[%.3f GB/s]\n", 1000.f*kernelDuration, toBW(totalBytes, kernelDuration));
    hipFree(device_in);
    hipFree(device_result);
    hipFree(out);
    hipFree(result_size);
    hipFree(histogram);
}

__device__ int get_index_to_check(int thread, int num_threads, int set_size, int offset) {

  // Integer division trick to round up
  return (((set_size + num_threads) / num_threads) * thread) + offset;
}

__device__ void search_lower(int search, int array_length,  int2 *arr, int *ret_val ) {
  const int num_threads = blockDim.x;
  const int thread = threadIdx.x;
  int set_size = array_length;
  while(set_size != 0){
    // Get the offset of the array, initially set to 0
    int offset = ret_val[1];

    // I think this is necessary in case a thread gets ahead, and resets offset before it's read
    // This isn't necessary for the unit tests to pass, but I still like it here
    __syncthreads();

    // Get the next index to check
    int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);

    // If the index is outside the bounds of the array then lets not check it
    if (index_to_check < array_length){
      // If the next index is outside the bounds of the array, then set it to maximum array size
      int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);

      if (next_index_to_check >= array_length){
        next_index_to_check = array_length - 1;
      }
   /* if( search == 5 && blockIdx.x == 1) {
        cuPrintf("index to check arr[%d] = %d , arr[%d] = %d \n", index_to_check,arr[index_to_check].x, next_index_to_check, arr[next_index_to_check].x);
    }*/

      // If we're at the mid section of the array reset the offset to this index
      if (search > arr[index_to_check].x && (search <= arr[next_index_to_check].x)) {
        ret_val[1] = index_to_check;
      }
      else if (search == arr[index_to_check].x) {
        // Set the return var if we hit it
       /* if(blockIdx.x == 1 && search == 5) {
            cuPrintf("find it at %d %d\n", index_to_check, ret_val[0]);
        }*/
        atomicMin(&ret_val[0], index_to_check);
      }
    }

    // Since this is a p-ary search divide by our total threads to get the next set size
    set_size = set_size / num_threads;

    // Sync up so no threads jump ahead and get a bad offset
    __syncthreads();
  }
}

__device__ void search_upper(int search, int array_length,  int2 *arr, int *ret_val ) {
  const int num_threads = blockDim.x;
  const int thread = threadIdx.x;
  int set_size = array_length;
  while(set_size != 0){
    // Get the offset of the array, initially set to 0
    int offset = ret_val[1];

    // I think this is necessary in case a thread gets ahead, and resets offset before it's read
    // This isn't necessary for the unit tests to pass, but I still like it here
    __syncthreads();

    // Get the next index to check
    int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);

    // If the index is outside the bounds of the array then lets not check it
    if (index_to_check < array_length){
      // If the next index is outside the bounds of the array, then set it to maximum array size
      int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);

      if (next_index_to_check >= array_length){
        next_index_to_check = array_length - 1;
      }
   /* if( search == 5 && blockIdx.x == 1) {
        cuPrintf("index to check arr[%d] = %d , arr[%d] = %d \n", index_to_check,arr[index_to_check].x, next_index_to_check, arr[next_index_to_check].x);
    }*/

      // If we're at the mid section of the array reset the offset to this index
      if (search > arr[index_to_check].x && (search <= arr[next_index_to_check].x)) {
        ret_val[1] = index_to_check;
      }
      else if (search == arr[index_to_check].x) {
        // Set the return var if we hit it
       /* if(blockIdx.x == 1 && search == 5) {
            cuPrintf("find it at %d %d\n", index_to_check, ret_val[0]);
        }*/
        atomicMax(&ret_val[0], index_to_check);
      }
    }

    // Since this is a p-ary search divide by our total threads to get the next set size
    set_size = set_size / num_threads;

    // Sync up so no threads jump ahead and get a bad offset
    __syncthreads();
  }
}
__global__ void p_ary_search(int search, int array_length,  int2 *arr, int *ret_val ) {

  const int num_threads = blockDim.x * gridDim.x;
  const int thread = blockIdx.x * blockDim.x + threadIdx.x;
  //ret_val[0] = -1;
  //ret_val[1] = 0;

  int set_size = array_length;

  while(set_size != 0){
    // Get the offset of the array, initially set to 0
    int offset = ret_val[1];

    // I think this is necessary in case a thread gets ahead, and resets offset before it's read
    // This isn't necessary for the unit tests to pass, but I still like it here
    __syncthreads();

    // Get the next index to check
    int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);
    // If the index is outside the bounds of the array then lets not check it
    if (index_to_check < array_length){
      // If the next index is outside the bounds of the array, then set it to maximum array size
      int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);

      if (next_index_to_check >= array_length){
        next_index_to_check = array_length - 1;
      }

      // If we're at the mid section of the array reset the offset to this index
      if (search > arr[index_to_check].x && (search < arr[next_index_to_check].x)) {
        ret_val[1] = index_to_check;
      }
      else if (search == arr[index_to_check].x) {
        // Set the return var if we hit it
        ret_val[0] = index_to_check;
      }
    }

    // Since this is a p-ary search divide by our total threads to get the next set size
    set_size = set_size / num_threads;

    // Sync up so no threads jump ahead and get a bad offset
    __syncthreads();
  }
}

__global__ void pnary_partition(int2* rel_a, int2* rel_b, int* lower_array, int* upper_array, float* out_bound, int N, int M) {
	const int lower_bound = rel_a[blockIdx.x *  blockDim.x].x;
   	const int upper_bound = rel_a[(blockIdx.x + 1) * blockDim.x - 1].x;
    __shared__ int lower;
    __shared__ int upper;
    lower_array[2 * blockIdx.x] = M;
    lower_array[2 * blockIdx.x + 1] = 0;
    upper_array[2 * blockIdx.x] = -1;
    upper_array[2 * blockIdx.x + 1] = 0;
    __syncthreads();
    search_lower(lower_bound, M, rel_b, &lower_array[2 * blockIdx.x]);
    search_upper(upper_bound, M, rel_b, &upper_array[2 * blockIdx.x]);
    lower = lower_array[2 * blockIdx.x] < M? lower_array[2 * blockIdx.x]:lower_array[2 * blockIdx.x + 1];
    upper = upper_array[2 * blockIdx.x] >= 0? upper_array[2 * blockIdx.x]:upper_array[2 * blockIdx.x + 1];
    if( upper < lower) {
        upper = M - 1;
    }
    out_bound[blockIdx.x] = blockDim.x * ( upper - lower + 1); 
    /*if(threadIdx.x == 0) {
    cuPrintf("lower_bound: %d ret: %d offset: %d\n", lower_bound, lower_array[2 * blockIdx.x], lower_array[2 * blockIdx.x + 1]);
    cuPrintf("upper_bound: %d ret: %d offset: %d\n", upper_bound, upper_array[2 * blockIdx.x], upper_array[2 * blockIdx.x + 1]);
    cuPrintf("num result tuples: %f\n", out_bound[blockIdx.x]);
    }*/
}

void
__global__ brute_join( int3* out, int2* rel_a, int2* rel_b, int num, int N, int M, float* out_bound, float* result_size, int* lower_array, int* upper_array ) {
    __shared__ int2 left[512];
    __shared__ int2 right[1024];
    __shared__ uint count[512];
    __shared__ uint index[512];
    __shared__ uint scratch[1024];
    int lower;
    int upper;
    int num_right;
    lower = lower_array[2 * blockIdx.x] < M? lower_array[2 * blockIdx.x]:lower_array[2 * blockIdx.x + 1];
    upper = upper_array[2 * blockIdx.x] >= 0? upper_array[2 * blockIdx.x]:upper_array[2 * blockIdx.x + 1];
    if( upper < lower) {
        upper = M - 1;
    }
    num_right = upper - lower + 1;
    int threadIndex =  threadIdx.x;
    int partition = blockIdx.x * blockDim.x;
    // counter for each thread
    count[threadIndex] = 0;
    index[threadIndex] = 0;
    // load two relation to the cache, make future access faster
    left[threadIndex] = rel_a[partition + threadIndex];
    for(int i = 0 ; i < num_right; i+= 512) {
        if(i + threadIndex < num_right) {
            //cuPrintf("%d\n",lower + i + threadIndex);
            right[i + threadIndex] = rel_b[lower + i + threadIndex];
        }
        __syncthreads();
   }
    for(int i = 0 ; i < num_right; i++ ) {
        if(left[threadIndex].x == right[i].x) {
            count[threadIndex] ++;
        }
    }
    __syncthreads();
    sharedMemExclusiveScan(threadIndex, count, index, scratch, SCAN_BLOCK_DIM);
    int current = 0;
    for(int i = 0 ; i < num_right; i++ ) {
        if(left[threadIndex].x == right[i].x) {
           int j = (int)out_bound[blockIdx.x] + index[threadIndex] + current;
           //cuPrintf("out index %d of %d\n", j, num);
           if( j < num) {
            out[j].x = left[threadIndex].x;
            out[j].y = left[threadIndex].y;
            out[j].z = right[i].y;
            if( blockIdx.x == 1) {
                //cuPrintf("%d\n",out[j].x);
               // cuPrintf("index %d =  %d + %d + %d\n", j, (int)out_bound[blockIdx.x], index[threadIndex], current);
            }
            current++;
           } 
        }
    }
    if(threadIdx.x == 0) {
        result_size[blockIdx.x] = count[511] + index[511];
        //cuPrintf("result size: %f\n",result_size[blockIdx.x]);
    }
}

void
__global__ join_coalesced(int3* result, int3* out, float* result_size, float* histogram, float* out_bound) {
    int size = result_size[blockIdx.x];
    int result_index;
    int out_index;
    for(int i = 0; i < size; i += 512) {
        if(i + threadIdx.x < size) {
           out_index = out_bound[blockIdx.x] + threadIdx.x + i;
           result_index = histogram[blockIdx.x] + threadIdx.x + i;
           if(blockIdx.x == 1) {
                //cuPrintf("## %d  = %d  + %d + %d\n", out_index, (int)out_bound[blockIdx.x], threadIdx.x, i);
           }
          /* if(result_index == 528) {
                cuPrintf("## %d\n", out[out_index].x);
                cuPrintf("## %d  = %d  + %d + %d\n", out_index, (int)out_bound[blockIdx.x], threadIdx.x, i);
           }*/
           result[result_index].x = out[out_index].x;
           result[result_index].y = out[out_index].y;
           result[result_index].z = out[out_index].z;
        }
        __syncthreads();
    }
}


/*
    Implementation of JOIN operationi
    rel_a: left relation
    rel_b: right relation
    N: size of rel_a
    M: size of rel_b
*/
struct compare_int2 {

    __host__ __device__
    bool operator()(int2 a, int2 b) {
        return a.x <= b.x;
    }
};

void primitive_join(int N, int M) {
    // prepare host buffers
    int min = 1;
    int max = 1024;
    int2* rel_a = new int2[N];
    int2* rel_b = new int2[M];
    int3* result_seq = new int3[4 * N];
    int seq_num;
    for(int i = 0; i < N; i ++) {
        rel_a[i] = make_int2(min + (rand() % (int)(max - min + 1)), min + (rand() % (int)(max - min + 1)) );
    }
    for(int i = 0; i < M; i ++) {
        rel_b[i] = make_int2(min + (rand() % (int)(max - min + 1)), min + (rand() % (int)(max - min + 1)) );
    }
    thrust::sort(rel_a, rel_a + N, compare_int2());
    thrust::sort(rel_b, rel_b + M, compare_int2());

    sequential_join(rel_a, rel_b, N, M, result_seq, &seq_num);

    // prepare device buffers
	const int threadPerBlock = 512;
	const int blocks = (N + threadPerBlock - 1) / threadPerBlock;
    printf("num blocks: %d\n", blocks);
    int2* dev_rel_a;
    int2* dev_rel_b;
    int* lower_array;
    int* upper_array;
    float* out_bound;
    //float* out_bound_scan;
    float* result_size;
    float* histogram;
    int3* out;
    int3* result;
    hipMalloc((void**) &out, sizeof(int3) * N * M);
    hipMalloc((void**) &result, sizeof(int3) * N * M);
    hipMalloc((void**) &result_size, sizeof(float) * blocks);
    hipMalloc((void**) &histogram, sizeof(float) * blocks);
    hipMalloc((void**) &out_bound, sizeof(float) * blocks);
    hipMalloc((void**) &lower_array, sizeof(int) * blocks * 2);
    hipMalloc((void**) &upper_array, sizeof(int) * blocks * 2);
    hipMalloc((void**) &dev_rel_a, sizeof(int2) * N);
    hipMalloc((void**) &dev_rel_b, sizeof(int2) * M);
	
    double startTime = CycleTimer::currentSeconds();
    hipMemcpy(dev_rel_a, rel_a, sizeof(int2) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_rel_b, rel_b, sizeof(int2) * M, hipMemcpyHostToDevice);
    cudaPrintfInit();

    double startTime_inner = CycleTimer::currentSeconds();

    pnary_partition<<< blocks, threadPerBlock >>>(dev_rel_a, dev_rel_b, lower_array, upper_array ,out_bound, N, M);
    thrust::device_ptr<float> dev_ptr1(out_bound);
    thrust::exclusive_scan(dev_ptr1, dev_ptr1 + blocks, dev_ptr1);
    //prescanArray(out_bound, out_bound, blocks);
    //deallocBlockSums();
    brute_join<<< blocks, threadPerBlock >>>(out, dev_rel_a, dev_rel_b,  N * M , N, M, out_bound, result_size, lower_array, upper_array);

    thrust::device_ptr<float> dev_ptr2(result_size);
    thrust::device_ptr<float> dev_ptr3(histogram);
    thrust::exclusive_scan(dev_ptr2, dev_ptr2 + blocks, dev_ptr3);
    join_coalesced<<<blocks, threadPerBlock>>>(result, out, result_size, histogram, out_bound); 
   
   double endTime_inner = CycleTimer::currentSeconds();
   printf("time excution from cuda join kernel %.3f ms\n",1000.f * (endTime_inner  - startTime_inner));

    float* p = new float[blocks];
    int numResult = 0; 
	hipMemcpy(p, result_size, sizeof(float) * blocks, hipMemcpyDeviceToHost);
    numResult += p[blocks-1];
	hipMemcpy(p, histogram, sizeof(float) * blocks, hipMemcpyDeviceToHost);
    numResult += p[blocks-1];
    printf("cuda produces %d tuples\n", numResult);

    int3* tmp_check = new int3[numResult];
	hipMemcpy(tmp_check, result, sizeof(int3)*numResult, hipMemcpyDeviceToHost);
    double endTime = CycleTimer::currentSeconds();
    printf("time excution from cuda join overall %.3f ms\n",1000.f * (endTime  - startTime));
    
    validate_join(result_seq, seq_num, tmp_check, numResult);

    cudaPrintfDisplay(stdout, true);
 	cudaPrintfEnd();
    hipFree(dev_rel_a);
    hipFree(dev_rel_b);
    hipFree(lower_array);
    hipFree(upper_array);
    hipFree(out_bound);
    hipFree(result_size);
    hipFree(out);
    hipFree(histogram);
    hipFree(result);
  //  deallocBlockSums();
}



#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel( int *a, int *b, int *c ) {
 int idx = threadIdx.x + blockIdx.x * blockDim.x;
      if (idx < N) {
             int idx1 = (idx + 1) % 256;
             int idx2 = (idx + 2) % 256;
             float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
             float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
             c[idx] = (as + bs) / 2;
        }
}

void  streamTest() {

  hipDeviceProp_t  prop;
    int whichDevice;
    HANDLE_ERROR( hipGetDevice( &whichDevice ) );
    HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream0, stream1;
    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;

    // start the timers
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    // initialize the streams
    HANDLE_ERROR( hipStreamCreate( &stream0 ) );
    HANDLE_ERROR( hipStreamCreate( &stream1 ) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a0,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b0,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c0,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_a1,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b1,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c1,
                              N * sizeof(int) ) );

    // allocate host locked memory, used to stream
    HANDLE_ERROR( hipHostAlloc( (void**)&host_a,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_b,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_c,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    // now loop over full data, in bite-sized chunks
    for (int i=0; i<FULL_DATA_SIZE; i+= N*2) {
        // enqueue copies of a in stream0 and stream1
        HANDLE_ERROR( hipMemcpyAsync( dev_a0, host_a+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream0 ) );
        HANDLE_ERROR( hipMemcpyAsync( dev_a1, host_a+i+N,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream1 ) );
        // enqueue copies of b in stream0 and stream1
        HANDLE_ERROR( hipMemcpyAsync( dev_b0, host_b+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream0 ) );
        HANDLE_ERROR( hipMemcpyAsync( dev_b1, host_b+i+N,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream1 ) );

        // enqueue kernels in stream0 and stream1
        kernel<<<N/256,256,0,stream0>>>( dev_a0, dev_b0, dev_c0 );
        kernel<<<N/256,256,0,stream1>>>( dev_a1, dev_b1, dev_c1 );

        // enqueue copies of c from device to locked memory
        HANDLE_ERROR( hipMemcpyAsync( host_c+i, dev_c0,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream0 ) );
        HANDLE_ERROR( hipMemcpyAsync( host_c+i+N, dev_c1,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream1 ) );
    }
    HANDLE_ERROR( hipStreamSynchronize( stream0 ) );
    HANDLE_ERROR( hipStreamSynchronize( stream1 ) );

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );

    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );

    // cleanup the streams and memory
    HANDLE_ERROR( hipHostFree( host_a ) );
    HANDLE_ERROR( hipHostFree( host_b ) );
    HANDLE_ERROR( hipHostFree( host_c ) );
    HANDLE_ERROR( hipFree( dev_a0 ) );
    HANDLE_ERROR( hipFree( dev_b0 ) );
    HANDLE_ERROR( hipFree( dev_c0 ) );
    HANDLE_ERROR( hipFree( dev_a1 ) );
    HANDLE_ERROR( hipFree( dev_b1 ) );
    HANDLE_ERROR( hipFree( dev_c1 ) );
    HANDLE_ERROR( hipStreamDestroy( stream0 ) );
    HANDLE_ERROR( hipStreamDestroy( stream1 ) );
}

void sequentialTest() {
    hipEvent_t     start, stop;
    float           elapsedTime;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    int* host_a = new int[FULL_DATA_SIZE];
    int* host_b = new int[FULL_DATA_SIZE];
    int* host_c = new int[FULL_DATA_SIZE];
    int *dev_a, *dev_b, *dev_c;
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, FULL_DATA_SIZE * sizeof(int));
    hipMalloc( (void**)&dev_b, FULL_DATA_SIZE * sizeof(int));
    hipMalloc( (void**)&dev_c, FULL_DATA_SIZE * sizeof(int));
    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }
    HANDLE_ERROR( hipEventRecord( start, 0 ) );
	hipMemcpy(dev_a, host_a, sizeof(int) * FULL_DATA_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * FULL_DATA_SIZE, hipMemcpyHostToDevice);
    kernel<<<FULL_DATA_SIZE/256,256,0>>>( dev_a, dev_b, dev_c);
    hipMemcpyAsync( host_c, dev_c, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );
}

void
printCudaInfo() {
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
}
