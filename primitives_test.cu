#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "hip/device_functions.h"
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "CycleTimer.h"
#define SCAN_BLOCK_DIM 512
#define uint unsigned int
#include "exclusiveScan.cu_inl"
#include "cuPrintf.cu"
#include "scan.cu"
#include "book.h"
#include "parallel_scan.cu"

#define GRID_DIM 65535

using namespace std;
extern float toBW(int bytes, float sec);


//brute force find matching tuples
void sequential_join(int2* rel_a, int2* rel_b, int rel_a_size, int rel_b_size, int3* out, int* numResult) {
   *numResult = 0;
   double startTime = CycleTimer::currentSeconds();
     for (int i = 0; i < rel_a_size; i++) {
         for(int j = 0; j < rel_b_size; j ++ ) {
            if(rel_a[i].x == rel_b[j].x) {
                out[*numResult].x = rel_a[i].x;
                out[*numResult].y = rel_a[i].y;
                out[*numResult].z = rel_b[j].y;
                (*numResult)++;
            }
            else if( rel_a[i].x < rel_b[j].x) {
                break;
            }
        }
     }
   double endTime = CycleTimer::currentSeconds();
   printf("time excution from sequential join %.3f ms\n",1000.f * (endTime  - startTime));
   printf("sequential join produces %d tuples\n", *numResult);
}

void validate_join(int3* seq_out, int seq_num, int3* cuda_out, int cuda_num) {
    if( seq_num != cuda_num ) {
        printf("num of tuples seq(%d) != cuda(%d)", seq_num, cuda_num);
        return;
    }
    for(int i = 0 ; i < cuda_num; i ++) {
       if( seq_out[i].x != cuda_out[i].x ||seq_out[i].y != cuda_out[i].y || seq_out[i].z != cuda_out[i].z) {
            printf("At line %d, not match FAIL\n",i);
            printf("seq: [%d, %d, %d], GPU: [%d, %d, %d]\n",seq_out[i].x,seq_out[i].y, seq_out[i].z, cuda_out[i].x, cuda_out[i].y, cuda_out[i].z);
            return;
       }
    }
    printf("JOIN PASS !\n");
}


/*
    choose the quilified tuples from the relation
    and get the cout of tuples of each block
*/
__global__ void
primitive_select_kernel(int N, int blocks, int* tuples, int* result, float* result_size) {

	__shared__ uint input[SCAN_BLOCK_DIM];
	__shared__ uint output[SCAN_BLOCK_DIM];
	__shared__ uint scratch[2 * SCAN_BLOCK_DIM];
	int threadIndex =  threadIdx.x;
	int partition = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    int blockIndex = blockIdx.y * gridDim.x + blockIdx.x;

    if (partition + threadIndex >= N || blockIndex >= blocks) {
        return;
    }

	//cuPrintf("%d %d %d\n", blockIdx.y, gridDim.x, blockIndex);
	input[threadIndex] = 0;
	output[threadIndex] = 0;
 	if ( partition + threadIndex < N ) {
		input[threadIndex] = tuples[partition + threadIndex] % 2 == 0? 1 : 0;
 	}
	 __syncthreads();
	 sharedMemExclusiveScan(threadIndex, input, output, scratch, SCAN_BLOCK_DIM);
	if(input[threadIndex]){
		 //atomicAdd(result_size + blockIndex, 1);
    	 result[partition + output[threadIndex]] = tuples[partition + threadIndex];
 	}
    /*for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset) {
          // add a partial sum upstream to our own
          input[threadIdx.x] += input[threadIdx.x + offset];
     }

        // wait until all threads in the block have
        // updated their partial sums
        __syncthreads();
      }*/
      // thread 0 writes the final result
      if(threadIdx.x == 0) {
          result_size[blockIndex] = input[511] + output[511];
      }
}

/*
    gather stage
*/
__global__ void coalesced(int N, int* result, float* result_size, float* histogram, int* out) {
	int threadIndex =  threadIdx.x;
	int partition = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;
    int blockIndex = blockIdx.y * gridDim.x + blockIdx.x;

    if (partition + threadIndex >= N) {
        return;
    }

    if( threadIndex < (int)result_size[blockIndex]) {
          /*if(histogram[blockIndex] + threadIndex == 0) {
            cuPrintf("%d %d\n", blockIndex, threadIndex);
          }*/
		  out[(int)histogram[blockIndex] + threadIndex] = result[partition + threadIndex];
	   }
	__syncthreads();
}

/*
    This is a sample of how to use scanLargeArray
    from Nvidia SDK
*/
void primitive_scan(int N, int inData[], int outData[]) {
	int large_num = 2048;
    float tmp[large_num];
    float* large_in;
    float* large_out;
    hipStream_t stream0;
    hipStreamCreate( &stream0 );
    double startTime;
    double endTime;
	hipMalloc((void**) &large_in, sizeof(float) * large_num);
	hipMalloc((void**) &large_out, sizeof(float) * large_num);
    //hipMemset(large_in, 1, large_num * sizeof(float));
    for(int i = 0; i < large_num; i ++) {
        tmp[i] = 1.0;
    }
	hipMemcpy(large_in, tmp, sizeof(float) * large_num, hipMemcpyHostToDevice);
    startTime = CycleTimer::currentSeconds();
    thrust::device_ptr<float> dev_ptr1(large_in);
    thrust::device_ptr<float> dev_ptr2(large_out);
    thrust::exclusive_scan(dev_ptr1, dev_ptr1 + large_num, dev_ptr2);
    endTime = CycleTimer::currentSeconds();
   printf("time excution from thrust scan %.3f ms\n",1000.f * (endTime  - startTime));
    startTime = CycleTimer::currentSeconds();
    scan_up<<<2, 512>>>(large_in, large_out);
    scan_sum<<<1, 1>>>(large_in, large_out);
    scan_down<<<2, 512>>>(large_in, large_out);
   //  preallocBlockSums(large_num);
   //  prescanArray(large_out, large_in, large_num, stream0);
    endTime = CycleTimer::currentSeconds();
    printf("time excution from parallel array scan %.3f ms\n", 1000.f * (endTime  - startTime));

    hipMemcpy(tmp, large_out, sizeof(float) * large_num, hipMemcpyDeviceToHost);
    for(int i = 0; i < large_num; i ++) {
        printf("%f ", tmp[i]);
    }
    printf("\n");
    int y[] = {1, 2};
    printf("%d\n", y[(int)tmp[1]]);
    deallocBlockSums();
}


/*
    Implementation of SELECT operation
*/
void
primitive_select(int N, int inData[], int outData[]) {
	const int threadPerBlock = 512;
	const int blocks = (N + threadPerBlock - 1) / threadPerBlock;

  int rows = (blocks / GRID_DIM) == 0? 1 : (blocks / GRID_DIM) + 1;
  int cols = (blocks / GRID_DIM) == 0? blocks : GRID_DIM;
  dim3 blockDim(threadPerBlock, 1);
  dim3 gridDim(cols, rows);
    printf("rows %d cols: %d\n", rows, cols);
	const int blocksOfReulstSize = ( blocks + threadPerBlock - 1) / threadPerBlock;
    int totalBytes = N * sizeof(int) * 2;
    printf("Num of tuples %d\n", N);
	printf("Num of blocks %d\n", blocks);
	printf("Num of blocks for result size %d\n", blocksOfReulstSize);
    int* device_in;
	int* device_result;
	float* result_size;
	float* histogram;
	int* out;
	int* tmp = (int*)calloc(N, sizeof(int));
	hipMalloc((void**) &device_in, sizeof(int) * N);
	hipMalloc((void**) &device_result, sizeof(int) * N);
	hipMalloc((void**) &out, sizeof(int) * N);
	hipMalloc((void**) &result_size, sizeof(float) * blocks);
	hipMalloc((void**) &histogram, sizeof(float) * blocks);
    double startTime = CycleTimer::currentSeconds();
	hipMemcpy(device_in, inData, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(device_result, tmp, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(out, tmp, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(result_size, tmp, sizeof(int) * blocks, hipMemcpyHostToDevice);
    cudaPrintfInit();
    double startTime_inner = CycleTimer::currentSeconds();
//	for(int i = 0 ; i < 10 ; i ++) {
    primitive_select_kernel<<<gridDim, blockDim>>>(N, blocks, device_in, device_result, result_size);

   /* int test_result_size[blocks];
    hipMemcpy(test_result_size, result_size, sizeof(int) * blocks, hipMemcpyDeviceToHost);
    for(int i = 0 ; i < blocks ; i ++) {
       printf("%d: %d, ",i ,test_result_size[i]);
    }
    printf("\n");
	hipDeviceSynchronize();*/

    thrust::device_ptr<float> dev_ptr1(result_size);
    thrust::device_ptr<float> dev_ptr2(histogram);
    thrust::exclusive_scan(dev_ptr1, dev_ptr1 + blocks, dev_ptr2);
   /* int test_histgram[blocks];
    hipMemcpy(test_histgram, histogram, sizeof(int)*blocks, hipMemcpyDeviceToHost);
    for(int i = 0 ; i < blocks; i ++) {
        printf("%d, ", test_histgram[i]);
    }
    printf("\n");*/
	coalesced<<<gridDim, blockDim>>>(N, device_result, result_size, histogram, out);
  //  }
    double endTime_inner = CycleTimer::currentSeconds();
    cudaPrintfDisplay(stdout, true);
 	  cudaPrintfEnd();

    hipMemcpy(outData, out, sizeof(int) * N, hipMemcpyDeviceToHost);
    double endTime = CycleTimer::currentSeconds();

    double overallDuration = endTime - startTime;
    double kernelDuration = endTime_inner - startTime_inner;
    printf("CUDA overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("CUDA execution time for kernel: %.3f ms\t\t[%.3f GB/s]\n", 1000.f*kernelDuration, toBW(totalBytes, kernelDuration));
    hipFree(device_in);
    hipFree(device_result);
    hipFree(out);
    hipFree(result_size);
    hipFree(histogram);
}




/*
    Implementation of SELECT operation with stream
*/
void
primitive_select_stream(int N, int inData[], int outData[]) {

  int full_data_size = N;
  int one_stripe = 1024*1024;
  hipStream_t stream0, stream1, stream2;
  
  const int threadPerBlock = 512;
  const int blocks = (one_stripe + threadPerBlock - 1) / threadPerBlock;
  int rows = (blocks / GRID_DIM) == 0? 1 : (blocks / GRID_DIM) + 1;
  int cols = (blocks / GRID_DIM) == 0? blocks : GRID_DIM;
  dim3 blockDim(threadPerBlock, 1);
  dim3 gridDim(cols, rows);


  printf("rows %d cols: %d\n", rows, cols);
  const int blocksOfReulstSize = ( blocks + threadPerBlock - 1) / threadPerBlock;
  int totalBytes = N * sizeof(int) * 2;
  printf("Num of tuples %d\n", N);
  printf("Num of blocks %d\n", blocks);
  printf("Num of blocks for result size %d\n", blocksOfReulstSize);
  
  int* tmp = (int*)calloc(N, sizeof(int));

  hipStreamCreate( &stream0 );
  hipStreamCreate( &stream1 );
  hipStreamCreate( &stream2 );

  int* device_in_0;
  int* device_in_1;
  int* device_in_2;

  int* device_result_0;
  int* device_result_1;
  int* device_result_2;

  float* result_size_0;
  float* result_size_1;
  float* result_size_2;

  float* histogram_0;
  float* histogram_1;
  float* histogram_2;

  int* out_0;
  int* out_1;
  int* out_2;

  hipMalloc((void**) &device_in_0, sizeof(int) * one_stripe);
  hipMalloc((void**) &device_in_1, sizeof(int) * one_stripe);
  hipMalloc((void**) &device_in_2, sizeof(int) * one_stripe);

  hipMalloc((void**) &device_result_0, sizeof(int) * one_stripe);
  hipMalloc((void**) &device_result_1, sizeof(int) * one_stripe);
  hipMalloc((void**) &device_result_2, sizeof(int) * one_stripe);

  hipMemcpy(device_result_0, tmp, sizeof(int) * one_stripe, hipMemcpyHostToDevice);
  hipMemcpy(device_result_1, tmp, sizeof(int) * one_stripe, hipMemcpyHostToDevice);
  hipMemcpy(device_result_2, tmp, sizeof(int) * one_stripe, hipMemcpyHostToDevice);

  hipMalloc((void**) &out_0, sizeof(int) * one_stripe);
  hipMalloc((void**) &out_1, sizeof(int) * one_stripe);
  hipMalloc((void**) &out_2, sizeof(int) * one_stripe);
  
  hipMemcpy(out_0, tmp, sizeof(int) * one_stripe, hipMemcpyHostToDevice);
  hipMemcpy(out_1, tmp, sizeof(int) * one_stripe, hipMemcpyHostToDevice);
  hipMemcpy(out_2, tmp, sizeof(int) * one_stripe, hipMemcpyHostToDevice);

  hipMalloc((void**) &result_size_0, sizeof(float) * blocks);
  hipMalloc((void**) &result_size_1, sizeof(float) * blocks);
  hipMalloc((void**) &result_size_2, sizeof(float) * blocks);


  hipMalloc((void**) &histogram_0, sizeof(float) * blocks);
  hipMalloc((void**) &histogram_1, sizeof(float) * blocks);
  hipMalloc((void**) &histogram_2, sizeof(float) * blocks);

  int *host_inData, *host_outData;

  hipHostAlloc( (void**)&host_inData, full_data_size * sizeof(int), hipHostMallocDefault);
  hipHostAlloc( (void**)&host_outData, full_data_size * sizeof(int), hipHostMallocDefault);
  
  memcpy(host_inData, inData, full_data_size * sizeof(int));
  memcpy(host_outData, tmp, full_data_size * sizeof(int));

  double startTime = CycleTimer::currentSeconds();
   for (int i = 0; i < full_data_size; i += 3 * one_stripe) {
        // enqueue copies of 
       hipMemcpyAsync( device_in_0, host_inData+i, sizeof(int) * one_stripe, hipMemcpyHostToDevice, stream0);
       primitive_select_kernel<<<gridDim, blockDim, 0, stream0 >>>(one_stripe, blocks, device_in_0, device_result_0, result_size_0);
       scan_up<<< 2, 512, 0, stream0>>>(result_size_0, histogram_0);
       scan_sum<<< 1, 1, 0, stream0>>>(result_size_0, histogram_0);
       scan_down<<< 2, 512, 0, stream0>>>(result_size_0, histogram_0);
       coalesced<<<gridDim, blockDim, 0, stream0>>>(one_stripe, device_result_0, result_size_0, histogram_0, out_0);
       hipMemcpyAsync(host_outData + i, out_0, sizeof(int) * one_stripe, hipMemcpyDeviceToHost, stream0);

       hipMemcpyAsync( device_in_1, host_inData + i + one_stripe, sizeof(int) * one_stripe, hipMemcpyHostToDevice, stream1);
       primitive_select_kernel<<<gridDim, blockDim, 0, stream1 >>>(one_stripe, blocks, device_in_1, device_result_1, result_size_1);
       scan_up<<< 2, 512, 0, stream1>>>(result_size_1, histogram_1);
       scan_sum<<< 1, 1, 0, stream1>>>(result_size_1, histogram_1);
       scan_down<<< 2, 512, 0, stream1>>>(result_size_1, histogram_1);
       coalesced<<<gridDim, blockDim, 0, stream1>>>(one_stripe, device_result_1, result_size_1, histogram_1, out_1);
       hipMemcpyAsync(host_outData + i + one_stripe, out_1, sizeof(int) * one_stripe, hipMemcpyDeviceToHost, stream1);
      
       hipMemcpyAsync( device_in_2, host_inData + i + 2 * one_stripe, sizeof(int) * one_stripe, hipMemcpyHostToDevice, stream2);
       primitive_select_kernel<<<gridDim, blockDim, 0, stream2 >>>(one_stripe, blocks, device_in_2, device_result_2, result_size_2);
       scan_up<<< 2, 512, 0, stream2>>>(result_size_2, histogram_2);
       scan_sum<<< 1, 1, 0, stream2>>>(result_size_2, histogram_2);
       scan_down<<< 2, 512, 0, stream2>>>(result_size_2, histogram_2);
       coalesced<<<gridDim, blockDim, 0, stream2>>>(one_stripe, device_result_2, result_size_2, histogram_2, out_2);
       hipMemcpyAsync(host_outData + i + 2 * one_stripe, out_2, sizeof(int) * one_stripe, hipMemcpyDeviceToHost, stream2);
    }
    hipStreamSynchronize( stream0 );
    hipStreamSynchronize( stream1 );
    hipStreamSynchronize( stream2 );

    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    printf("CUDA SELECT overall with stream: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    
    memcpy(outData, host_outData, full_data_size * sizeof(int));
    int count = 0; 
    for(int i = 0; i < full_data_size; i++) {
        if(outData[i] != 0 && outData[i] % 2 == 0) {
            //printf("%d ", outData[i]);
            count++;
        }
    }
    //printf("\nGPU num %d\n", count);

    hipFree(device_in_0);
    hipFree(device_result_0);
    hipFree(out_0);
    hipFree(result_size_0);
    hipFree(histogram_0);
    
    hipFree(device_in_1);
    hipFree(device_result_1);
    hipFree(out_1);
    hipFree(result_size_1);
    hipFree(histogram_1);

    hipFree(device_in_2);
    hipFree(device_result_2);
    hipFree(out_2);
    hipFree(result_size_2);
    hipFree(histogram_2);

    hipHostFree(host_outData);
    hipHostFree(host_inData);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}


__device__ int get_index_to_check(int thread, int num_threads, int set_size, int offset) {

  // Integer division trick to round up
  return (((set_size + num_threads) / num_threads) * thread) + offset;
}

__device__ void search_lower(int search, int array_length,  int2 *arr, int *ret_val ) {
  const int num_threads = blockDim.x;
  const int thread = threadIdx.x;
  int set_size = array_length;
  while(set_size != 0){
    // Get the offset of the array, initially set to 0
    int offset = ret_val[1];

    // I think this is necessary in case a thread gets ahead, and resets offset before it's read
    // This isn't necessary for the unit tests to pass, but I still like it here
    __syncthreads();

    // Get the next index to check
    int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);

    // If the index is outside the bounds of the array then lets not check it
    if (index_to_check < array_length){
      // If the next index is outside the bounds of the array, then set it to maximum array size
      int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);

      if (next_index_to_check >= array_length){
        next_index_to_check = array_length - 1;
      }
   /* if( search == 5 && blockIdx.x == 1) {
        cuPrintf("index to check arr[%d] = %d , arr[%d] = %d \n", index_to_check,arr[index_to_check].x, next_index_to_check, arr[next_index_to_check].x);
    }*/

      // If we're at the mid section of the array reset the offset to this index
      if (search > arr[index_to_check].x && (search <= arr[next_index_to_check].x)) {
        ret_val[1] = index_to_check;
      }
      else if (search == arr[index_to_check].x) {
        // Set the return var if we hit it
       /* if(blockIdx.x == 1 && search == 5) {
            cuPrintf("find it at %d %d\n", index_to_check, ret_val[0]);
        }*/
        atomicMin(&ret_val[0], index_to_check);
      }
    }

    // Since this is a p-ary search divide by our total threads to get the next set size
    set_size = set_size / num_threads;

    // Sync up so no threads jump ahead and get a bad offset
    __syncthreads();
  }
}

__device__ void search_upper(int search, int array_length,  int2 *arr, int *ret_val ) {
  const int num_threads = blockDim.x;
  const int thread = threadIdx.x;
  int set_size = array_length;
  while(set_size != 0){
    // Get the offset of the array, initially set to 0
    int offset = ret_val[1];

    // I think this is necessary in case a thread gets ahead, and resets offset before it's read
    // This isn't necessary for the unit tests to pass, but I still like it here
    __syncthreads();

    // Get the next index to check
    int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);

    // If the index is outside the bounds of the array then lets not check it
    if (index_to_check < array_length){
      // If the next index is outside the bounds of the array, then set it to maximum array size
      int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);

      if (next_index_to_check >= array_length){
        next_index_to_check = array_length - 1;
      }
   /* if( search == 5 && blockIdx.x == 1) {
        cuPrintf("index to check arr[%d] = %d , arr[%d] = %d \n", index_to_check,arr[index_to_check].x, next_index_to_check, arr[next_index_to_check].x);
    }*/

      // If we're at the mid section of the array reset the offset to this index
      if (search > arr[index_to_check].x && (search <= arr[next_index_to_check].x)) {
        ret_val[1] = index_to_check;
      }
      else if (search == arr[index_to_check].x) {
        // Set the return var if we hit it
       /* if(blockIdx.x == 1 && search == 5) {
            cuPrintf("find it at %d %d\n", index_to_check, ret_val[0]);
        }*/
        atomicMax(&ret_val[0], index_to_check);
      }
    }

    // Since this is a p-ary search divide by our total threads to get the next set size
    set_size = set_size / num_threads;

    // Sync up so no threads jump ahead and get a bad offset
    __syncthreads();
  }
}
__global__ void p_ary_search(int search, int array_length,  int2 *arr, int *ret_val ) {

  const int num_threads = blockDim.x * gridDim.x;
  const int thread = blockIdx.x * blockDim.x + threadIdx.x;
  //ret_val[0] = -1;
  //ret_val[1] = 0;

  int set_size = array_length;

  while(set_size != 0){
    // Get the offset of the array, initially set to 0
    int offset = ret_val[1];

    // I think this is necessary in case a thread gets ahead, and resets offset before it's read
    // This isn't necessary for the unit tests to pass, but I still like it here
    __syncthreads();

    // Get the next index to check
    int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);
    // If the index is outside the bounds of the array then lets not check it
    if (index_to_check < array_length){
      // If the next index is outside the bounds of the array, then set it to maximum array size
      int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);

      if (next_index_to_check >= array_length){
        next_index_to_check = array_length - 1;
      }

      // If we're at the mid section of the array reset the offset to this index
      if (search > arr[index_to_check].x && (search < arr[next_index_to_check].x)) {
        ret_val[1] = index_to_check;
      }
      else if (search == arr[index_to_check].x) {
        // Set the return var if we hit it
        ret_val[0] = index_to_check;
      }
    }

    // Since this is a p-ary search divide by our total threads to get the next set size
    set_size = set_size / num_threads;

    // Sync up so no threads jump ahead and get a bad offset
    __syncthreads();
  }
}

__global__ void pnary_partition(int2* rel_a, int2* rel_b, int* lower_array, int* upper_array, float* out_bound, int N, int M) {
	const int lower_bound = rel_a[blockIdx.x *  blockDim.x].x;
   	const int upper_bound = rel_a[(blockIdx.x + 1) * blockDim.x - 1].x;
    __shared__ int lower;
    __shared__ int upper;
    lower_array[2 * blockIdx.x] = M;
    lower_array[2 * blockIdx.x + 1] = 0;
    upper_array[2 * blockIdx.x] = -1;
    upper_array[2 * blockIdx.x + 1] = 0;
    __syncthreads();
    search_lower(lower_bound, M, rel_b, &lower_array[2 * blockIdx.x]);
    search_upper(upper_bound, M, rel_b, &upper_array[2 * blockIdx.x]);
    lower = lower_array[2 * blockIdx.x] < M? lower_array[2 * blockIdx.x]:lower_array[2 * blockIdx.x + 1];
    upper = upper_array[2 * blockIdx.x] >= 0? upper_array[2 * blockIdx.x]:upper_array[2 * blockIdx.x + 1];
    if( upper < lower) {
        upper = M - 1;
    }
    out_bound[blockIdx.x] = blockDim.x * ( upper - lower + 1); 
    /*if(threadIdx.x == 0) {
    cuPrintf("lower_bound: %d ret: %d offset: %d\n", lower_bound, lower_array[2 * blockIdx.x], lower_array[2 * blockIdx.x + 1]);
    cuPrintf("upper_bound: %d ret: %d offset: %d\n", upper_bound, upper_array[2 * blockIdx.x], upper_array[2 * blockIdx.x + 1]);
    cuPrintf("num result tuples: %f\n", out_bound[blockIdx.x]);
    }*/
}

void
__global__ brute_join( int3* out, int2* rel_a, int2* rel_b, int num, int N, int M, float* out_bound, float* result_size, int* lower_array, int* upper_array ) {
    __shared__ int2 left[512];
    __shared__ int2 right[1024];
    __shared__ uint count[512];
    __shared__ uint index[512];
    __shared__ uint scratch[1024];
    int lower;
    int upper;
    int num_right;
    lower = lower_array[2 * blockIdx.x] < M? lower_array[2 * blockIdx.x]:lower_array[2 * blockIdx.x + 1];
    upper = upper_array[2 * blockIdx.x] >= 0? upper_array[2 * blockIdx.x]:upper_array[2 * blockIdx.x + 1];
    if( upper < lower) {
        upper = M - 1;
    }
    num_right = upper - lower + 1;
    int threadIndex =  threadIdx.x;
    int partition = blockIdx.x * blockDim.x;
    // counter for each thread
    count[threadIndex] = 0;
    index[threadIndex] = 0;
    // load two relation to the cache, make future access faster
    left[threadIndex] = rel_a[partition + threadIndex];
    for(int i = 0 ; i < num_right; i+= 512) {
        if(i + threadIndex < num_right) {
            //cuPrintf("%d\n",lower + i + threadIndex);
            right[i + threadIndex] = rel_b[lower + i + threadIndex];
        }
        __syncthreads();
   }
    for(int i = 0 ; i < num_right; i++ ) {
        if(left[threadIndex].x == right[i].x) {
            count[threadIndex] ++;
        }
    }
    __syncthreads();
    sharedMemExclusiveScan(threadIndex, count, index, scratch, SCAN_BLOCK_DIM);
    int current = 0;
    for(int i = 0 ; i < num_right; i++ ) {
        if(left[threadIndex].x == right[i].x) {
           int j = (int)out_bound[blockIdx.x] + index[threadIndex] + current;
           //cuPrintf("out index %d of %d\n", j, num);
           if( j < num) {
            out[j].x = left[threadIndex].x;
            out[j].y = left[threadIndex].y;
            out[j].z = right[i].y;
            if( blockIdx.x == 1) {
                //cuPrintf("%d\n",out[j].x);
               // cuPrintf("index %d =  %d + %d + %d\n", j, (int)out_bound[blockIdx.x], index[threadIndex], current);
            }
            current++;
           } 
        }
    }
    if(threadIdx.x == 0) {
        result_size[blockIdx.x] = count[511] + index[511];
        //cuPrintf("result size: %f\n",result_size[blockIdx.x]);
    }
}

void
__global__ join_coalesced(int3* result, int3* out, float* result_size, float* histogram, float* out_bound) {
    int size = result_size[blockIdx.x];
    int result_index;
    int out_index;
    for(int i = 0; i < size; i += 512) {
        if(i + threadIdx.x < size) {
           out_index = out_bound[blockIdx.x] + threadIdx.x + i;
           result_index = histogram[blockIdx.x] + threadIdx.x + i;
           if(blockIdx.x == 1) {
                //cuPrintf("## %d  = %d  + %d + %d\n", out_index, (int)out_bound[blockIdx.x], threadIdx.x, i);
           }
          /* if(result_index == 528) {
                cuPrintf("## %d\n", out[out_index].x);
                cuPrintf("## %d  = %d  + %d + %d\n", out_index, (int)out_bound[blockIdx.x], threadIdx.x, i);
           }*/
           result[result_index].x = out[out_index].x;
           result[result_index].y = out[out_index].y;
           result[result_index].z = out[out_index].z;
        }
        __syncthreads();
    }
}


/*
    Implementation of JOIN operationi
    rel_a: left relation
    rel_b: right relation
    N: size of rel_a
    M: size of rel_b
*/
struct compare_int2 {

    __host__ __device__
    bool operator()(int2 a, int2 b) {
        return a.x <= b.x;
    }
};

void primitive_join(int N, int M) {
    // prepare host buffers
    int min = 1;
    int max = 1024;
    int2* rel_a = new int2[N];
    int2* rel_b = new int2[M];
    int3* result_seq = new int3[4 * N];
    int seq_num;
    for(int i = 0; i < N; i ++) {
        rel_a[i] = make_int2(min + (rand() % (int)(max - min + 1)), min + (rand() % (int)(max - min + 1)) );
    }
    for(int i = 0; i < M; i ++) {
        rel_b[i] = make_int2(min + (rand() % (int)(max - min + 1)), min + (rand() % (int)(max - min + 1)) );
    }
    thrust::sort(rel_a, rel_a + N, compare_int2());
    thrust::sort(rel_b, rel_b + M, compare_int2());

    sequential_join(rel_a, rel_b, N, M, result_seq, &seq_num);

    // prepare device buffers
	  const int threadPerBlock = 512;
	  const int blocks = (N + threadPerBlock - 1) / threadPerBlock;
    printf("num blocks: %d\n", blocks);
    int2* dev_rel_a;
    int2* dev_rel_b;
    int* lower_array;
    int* upper_array;
    float* out_bound;
    //float* out_bound_scan;
    float* result_size;
    float* histogram;
    int3* out;
    int3* result;
    hipMalloc((void**) &out, sizeof(int3) * N * M);
    hipMalloc((void**) &result, sizeof(int3) * N * M);
    hipMalloc((void**) &result_size, sizeof(float) * blocks);
    hipMalloc((void**) &histogram, sizeof(float) * blocks);
    hipMalloc((void**) &out_bound, sizeof(float) * blocks);
    hipMalloc((void**) &lower_array, sizeof(int) * blocks * 2);
    hipMalloc((void**) &upper_array, sizeof(int) * blocks * 2);
    hipMalloc((void**) &dev_rel_a, sizeof(int2) * N);
    hipMalloc((void**) &dev_rel_b, sizeof(int2) * M);
	
    double startTime = CycleTimer::currentSeconds();
    hipMemcpy(dev_rel_a, rel_a, sizeof(int2) * N, hipMemcpyHostToDevice);
	  hipMemcpy(dev_rel_b, rel_b, sizeof(int2) * M, hipMemcpyHostToDevice);
    cudaPrintfInit();

    double startTime_inner = CycleTimer::currentSeconds();

    pnary_partition<<< blocks, threadPerBlock >>>(dev_rel_a, dev_rel_b, lower_array, upper_array , out_bound, N, M);
    thrust::device_ptr<float> dev_ptr1(out_bound);
    thrust::exclusive_scan(dev_ptr1, dev_ptr1 + blocks, dev_ptr1);
    //prescanArray(out_bound, out_bound, blocks);
    //deallocBlockSums();
    brute_join<<< blocks, threadPerBlock >>>(out, dev_rel_a, dev_rel_b,  N * M , N, M, out_bound, result_size, lower_array, upper_array);

    thrust::device_ptr<float> dev_ptr2(result_size);
    thrust::device_ptr<float> dev_ptr3(histogram);
    thrust::exclusive_scan(dev_ptr2, dev_ptr2 + blocks, dev_ptr3);
    join_coalesced<<<blocks, threadPerBlock>>>(result, out, result_size, histogram, out_bound); 
   
   double endTime_inner = CycleTimer::currentSeconds();
   printf("time excution from cuda join kernel %.3f ms\n",1000.f * (endTime_inner  - startTime_inner));

    float* p = new float[blocks];
    int numResult = 0; 
	hipMemcpy(p, result_size, sizeof(float) * blocks, hipMemcpyDeviceToHost);
    numResult += p[blocks-1];
	hipMemcpy(p, histogram, sizeof(float) * blocks, hipMemcpyDeviceToHost);
    numResult += p[blocks-1];
    printf("cuda produces %d tuples\n", numResult);

    int3* tmp_check = new int3[numResult];
	hipMemcpy(tmp_check, result, sizeof(int3)*numResult, hipMemcpyDeviceToHost);
    double endTime = CycleTimer::currentSeconds();
    printf("time excution from cuda join overall %.3f ms\n",1000.f * (endTime  - startTime));
    
    validate_join(result_seq, seq_num, tmp_check, numResult);

    cudaPrintfDisplay(stdout, true);
 	cudaPrintfEnd();
    hipFree(dev_rel_a);
    hipFree(dev_rel_b);
    hipFree(lower_array);
    hipFree(upper_array);
    hipFree(out_bound);
    hipFree(result_size);
    hipFree(out);
    hipFree(histogram);
    hipFree(result);
  //  deallocBlockSums();
}



#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel( int *a, int *b, int *c ) {
 int idx = threadIdx.x + blockIdx.x * blockDim.x;
      if (idx < N) {
             int idx1 = (idx + 1) % 256;
             int idx2 = (idx + 2) % 256;
             float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
             float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
             c[idx] = (as + bs) / 2;
        }
}

void  streamTest() {

  hipDeviceProp_t  prop;
    int whichDevice;
    HANDLE_ERROR( hipGetDevice( &whichDevice ) );
    HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream0, stream1;
    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;

    // start the timers
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    // initialize the streams
    HANDLE_ERROR( hipStreamCreate( &stream0 ) );
    HANDLE_ERROR( hipStreamCreate( &stream1 ) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a0,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b0,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c0,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_a1,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b1,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c1,
                              N * sizeof(int) ) );

    // allocate host locked memory, used to stream
    HANDLE_ERROR( hipHostAlloc( (void**)&host_a,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_b,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_c,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    // now loop over full data, in bite-sized chunks
    for (int i=0; i<FULL_DATA_SIZE; i+= N*2) {
        // enqueue copies of a in stream0 and stream1
        HANDLE_ERROR( hipMemcpyAsync( dev_a0, host_a+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream0 ) );
        HANDLE_ERROR( hipMemcpyAsync( dev_a1, host_a+i+N,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream1 ) );
        // enqueue copies of b in stream0 and stream1
        HANDLE_ERROR( hipMemcpyAsync( dev_b0, host_b+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream0 ) );
        HANDLE_ERROR( hipMemcpyAsync( dev_b1, host_b+i+N,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream1 ) );

        // enqueue kernels in stream0 and stream1
        kernel<<<N/256,256,0,stream0>>>( dev_a0, dev_b0, dev_c0 );
        kernel<<<N/256,256,0,stream1>>>( dev_a1, dev_b1, dev_c1 );

        // enqueue copies of c from device to locked memory
        HANDLE_ERROR( hipMemcpyAsync( host_c+i, dev_c0,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream0 ) );
        HANDLE_ERROR( hipMemcpyAsync( host_c+i+N, dev_c1,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream1 ) );
    }
    HANDLE_ERROR( hipStreamSynchronize( stream0 ) );
    HANDLE_ERROR( hipStreamSynchronize( stream1 ) );

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );

    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );

    // cleanup the streams and memory
    HANDLE_ERROR( hipHostFree( host_a ) );
    HANDLE_ERROR( hipHostFree( host_b ) );
    HANDLE_ERROR( hipHostFree( host_c ) );
    HANDLE_ERROR( hipFree( dev_a0 ) );
    HANDLE_ERROR( hipFree( dev_b0 ) );
    HANDLE_ERROR( hipFree( dev_c0 ) );
    HANDLE_ERROR( hipFree( dev_a1 ) );
    HANDLE_ERROR( hipFree( dev_b1 ) );
    HANDLE_ERROR( hipFree( dev_c1 ) );
    HANDLE_ERROR( hipStreamDestroy( stream0 ) );
    HANDLE_ERROR( hipStreamDestroy( stream1 ) );
}

void sequentialTest() {
    hipEvent_t     start, stop;
    float           elapsedTime;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    int* host_a = new int[FULL_DATA_SIZE];
    int* host_b = new int[FULL_DATA_SIZE];
    int* host_c = new int[FULL_DATA_SIZE];
    int *dev_a, *dev_b, *dev_c;
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, FULL_DATA_SIZE * sizeof(int));
    hipMalloc( (void**)&dev_b, FULL_DATA_SIZE * sizeof(int));
    hipMalloc( (void**)&dev_c, FULL_DATA_SIZE * sizeof(int));
    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }
    HANDLE_ERROR( hipEventRecord( start, 0 ) );
	hipMemcpy(dev_a, host_a, sizeof(int) * FULL_DATA_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * FULL_DATA_SIZE, hipMemcpyHostToDevice);
    kernel<<<FULL_DATA_SIZE/256,256,0>>>( dev_a, dev_b, dev_c);
    hipMemcpyAsync( host_c, dev_c, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );
}

void
printCudaInfo() {
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
}
