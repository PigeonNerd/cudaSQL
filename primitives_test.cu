#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "hip/device_functions.h"
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "CycleTimer.h"
#define SCAN_BLOCK_DIM 512
#define uint unsigned int
#include "exclusiveScan.cu_inl"
#include "cuPrintf.cu"
#include "scan.cu"
#include "book.h"
using namespace std;
extern float toBW(int bytes, float sec);


// This scan only work on small buffer, do not used on large array
__global__ void prescan(int *g_odata, int *g_idata, int n){

    extern __shared__ int temp[];// allocated on invocation
    int thid = threadIdx.x;
    int offset = 1;
    temp[2*thid] = g_idata[2*thid]; // load input into shared memory
    temp[2*thid+1] = g_idata[2*thid+1];
    for (int d = n>>1; d > 0; d >>= 1){ // build sum in place up the tree
        __syncthreads();
        if (thid < d){

            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    if (thid == 0) { temp[n - 1] = 0; } // clear the last element
    for (int d = 1; d < n; d *= 2){ // traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (thid < d){
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    g_odata[2*thid] = temp[2*thid]; // write results to device memory
    g_odata[2*thid+1] = temp[2*thid+1];
}

/*
    choose the quilified tuples from the relation
    and get the cout of tuples of each block
*/
__global__ void
primitive_select_kernel(int N, int* tuples, int* result, int* result_size) {

	__shared__ uint input[SCAN_BLOCK_DIM];
	__shared__ uint output[SCAN_BLOCK_DIM];
	__shared__ uint scratch[2 * SCAN_BLOCK_DIM];

	int threadIndex =  threadIdx.x;
	int partition = blockIdx.x *  blockDim.x;
	//cuPrintf("%d\n", threadIndex);
	input[threadIndex] = 0;
	output[threadIndex] = 0;
 	if ( partition + threadIndex < N ) {
		input[threadIndex] = tuples[partition + threadIndex] % 2 == 0? 1 : 0;
 	}
	 __syncthreads();
	 sharedMemExclusiveScan(threadIndex, input, output, scratch, SCAN_BLOCK_DIM);
	if(input[threadIndex]){
		 //atomicAdd(result_size + blockIdx.x, 1);
    	 result[partition + output[threadIndex]] = tuples[partition + threadIndex];
 	}

      for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset) {
          // add a partial sum upstream to our own
          input[threadIdx.x] += input[threadIdx.x + offset];
        }

        // wait until all threads in the block have
        // updated their partial sums
        __syncthreads();
      }

      // thread 0 writes the final result
      if(threadIdx.x == 0) {
        result_size[blockIdx.x] = input[0];
      }
}

/*
    gather stage
*/
__global__ void coalesced(int N, int* result, int* result_size, int* histogram, int* out) {
	int threadIndex =  threadIdx.x;
	int partition = blockIdx.x *  blockDim.x;
    if( threadIndex < result_size[blockIdx.x] ) {
		out[histogram[blockIdx.x] + threadIndex] = result[partition + threadIndex];
	}
	__syncthreads();
}


/*
    This is a sample of how to use scanLargeArray
    from Nvidia SDK
*/
void
primitive_scan(int N, int inData[], int outData[]) {
	int large_num = 39063;
    float tmp[large_num];
    float* large_in;
    float* large_out;
    double startTime;
    double endTime;
	hipMalloc((void**) &large_in, sizeof(float) * large_num);
	hipMalloc((void**) &large_out, sizeof(float) * large_num);
    //hipMemset(large_in, 1, large_num * sizeof(float));
    for(int i = 0; i < large_num; i ++) {
        tmp[i] = 1.0;
    }
	hipMemcpy(large_in, tmp, sizeof(float) * large_num, hipMemcpyHostToDevice);
    startTime = CycleTimer::currentSeconds();
    preallocBlockSums(large_num);
    prescanArray(large_out, large_in, large_num);
    endTime = CycleTimer::currentSeconds();
   printf("time excution from large array scan %.3f ms\n", 1000.f * (endTime  - startTime));
   /* startTime = CycleTimer::currentSeconds();
    thrust::device_ptr<float> dev_ptr1(large_in);
    thrust::device_ptr<float> dev_ptr2(large_out);
    thrust::exclusive_scan(dev_ptr1, dev_ptr1 + large_num, dev_ptr2);
    endTime = CycleTimer::currentSeconds();
   printf("time excution from thrust scan %.3f ms\n",1000.f * (endTime  - startTime));*/
    hipMemcpy(tmp, large_out, sizeof(float) * large_num, hipMemcpyDeviceToHost);
    for(int i = 0; i < large_num; i ++) {
        printf("%f ", tmp[i]);
    }
    printf("\n");
    int y[] = {1, 2};
    printf("%d\n", y[(int)tmp[1]]);
}


/*
    Implementation of SELECT operation
*/
void
primitive_select(int N, int inData[], int outData[]) {
	const int threadPerBlock = 512;
	const int blocks = (N + threadPerBlock - 1) / threadPerBlock;
	const int blocksOfReulstSize = ( blocks + threadPerBlock - 1) / threadPerBlock;
    int totalBytes = N * sizeof(int) * 2;
    printf("Num of tuples %d\n", N);
	printf("Num of blocks %d\n", blocks);
	printf("Num of blocks for result size %d\n", blocksOfReulstSize);
    int* device_in;
	int* device_result;
	int* result_size;
	int* histogram;
	int* out;
	int* tmp = (int*)calloc(N, sizeof(int));
    double startTime = CycleTimer::currentSeconds();
	hipMalloc((void**) &device_in, sizeof(int) * N);
	hipMalloc((void**) &device_result, sizeof(int) * N);
	hipMalloc((void**) &out, sizeof(int) * N);
	hipMalloc((void**) &result_size, sizeof(int) * blocks);
	hipMalloc((void**) &histogram, sizeof(int) * blocks);
	hipMemcpy(device_in, inData, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(device_result, tmp, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(out, tmp, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(result_size, tmp, sizeof(int) * blocks, hipMemcpyHostToDevice);
    cudaPrintfInit();
    double startTime_inner = CycleTimer::currentSeconds();
	for(int i = 0 ; i < 10 ; i ++) {
    primitive_select_kernel<<<blocks, threadPerBlock>>>(N, device_in, device_result, result_size);

   // int test_result_size[blocks];
   // hipMemcpy(test_result_size, result_size, sizeof(int)*blocks, hipMemcpyDeviceToHost);
   // for(int i = 0 ; i < blocks ; i ++) {
   //     printf("%d, ", test_result_size[i]);
   // }
   // printf("\n");
	hipDeviceSynchronize();
	//prescan<<<blocksOfReulstSize, threadPerBlock, blocks * threadPerBlock * 2 * sizeof(int)>>>(histogram, result_size, blocks);

    thrust::device_ptr<int> dev_ptr1(result_size);
    thrust::device_ptr<int> dev_ptr2(histogram);
    thrust::exclusive_scan(dev_ptr1, dev_ptr1 + blocks, dev_ptr2);
   // int test_histgram[blocks];
   // hipMemcpy(test_histgram, histogram, sizeof(int)*blocks, hipMemcpyDeviceToHost);
   // for(int i = 0 ; i < blocks; i ++) {
   //     printf("%d, ", test_histgram[i]);
   // }
   // printf("\n");
	coalesced<<<blocks, threadPerBlock>>>(N, device_result, result_size, histogram, out);
    }
    double endTime_inner = CycleTimer::currentSeconds();
    cudaPrintfDisplay(stdout, true);
 	cudaPrintfEnd();
    hipMemcpy(outData, out, sizeof(int) * N, hipMemcpyDeviceToHost);
    double endTime = CycleTimer::currentSeconds();

    double overallDuration = endTime - startTime;
    double kernelDuration = endTime_inner - startTime_inner;
    printf("CUDA overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    printf("CUDA execution time for kernel: %.3f ms\t\t[%.3f GB/s]\n", 1000.f*kernelDuration, toBW(totalBytes, kernelDuration));
    hipFree(device_in);
    hipFree(device_result);
    hipFree(out);
    hipFree(result_size);
    hipFree(histogram);
}

__device__ int get_index_to_check(int thread, int num_threads, int set_size, int offset) {
 
  // Integer division trick to round up
  return (((set_size + num_threads) / num_threads) * thread) + offset;
}


__global__ void p_ary_search(int search, int array_length,  int *arr, int *ret_val ) {
 
  const int num_threads = blockDim.x * gridDim.x;
  const int thread = blockIdx.x * blockDim.x + threadIdx.x;
  
  //ret_val[0] = -1;
  //ret_val[1] = offset;
 
  int set_size = array_length;
 
  
  while(set_size != 0){
    // Get the offset of the array, initially set to 0
    int offset = ret_val[1];
    
    // I think this is necessary in case a thread gets ahead, and resets offset before it's read
    // This isn't necessary for the unit tests to pass, but I still like it here
    __syncthreads();  
 
    // Get the next index to check
    int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);
 
    // If the index is outside the bounds of the array then lets not check it
    if (index_to_check < array_length){
 
      // If the next index is outside the bounds of the array, then set it to maximum array size
      int next_index_to_check = get_index_to_check(thread + 1, num_threads, set_size, offset);
 
      if (next_index_to_check >= array_length){
        next_index_to_check = array_length - 1;
      }
 
      // If we're at the mid section of the array reset the offset to this index
      if (search > arr[index_to_check] && (search < arr[next_index_to_check])) {
        ret_val[1] = index_to_check;
      }
      else if (search == arr[index_to_check]) {
        // Set the return var if we hit it
        ret_val[0] = index_to_check;
      } 
    }
 
    // Since this is a p-ary search divide by our total threads to get the next set size
    set_size = set_size / num_threads;
    
    // Sync up so no threads jump ahead and get a bad offset
    __syncthreads();
  }
}

__global__ void binary_partition(int2* rel_a, int2* rel_b, int* out_bound, int N, int M) {
	int threadIndex =  threadIdx.x;
	int partition = blockIdx.x *  blockDim.x;
	const int lower_bound = rel_a[blockIdx.x *  blockDim.x].x;
   	const int upper_bound = rel_a[(blockIdx.x + 1) * blockDim.x - 1].x;

	int low_index = binary_search(rel_b, lower_bound, 0, M);
	int high_index = binary_search(rel_b, upper_bound, 0, M);

	__syncthreads();

	//prefix sum of outbound after finish all blocks of rel_a


	//JOIN

}


/*
    Implementation of JOIN operationi
    rel_a: left relation
    rel_b: right relation
    N: size of rel_a
    M: size of rel_b
*/
struct compare_int2 {

    __host__ __device__
    bool operator()(int2 a, int2 b) {
        return a.x <= b.x;
    }
};

void primitive_join(int N, int M) {
    // prepare host buffers
    int min = 1;
    int max = 20;
    int2* rel_a = new int2[N];
    int2* rel_b = new int2[M];
    for(int i = 0; i < N; i ++) {
        rel_a[i] = make_int2(min + (rand() % (int)(max - min + 1)), min + (rand() % (int)(max - min + 1)) );
    }
    for(int i = 0; i < M; i ++) {
        rel_b[i] = make_int2(min + (rand() % (int)(max - min + 1)), min + (rand() % (int)(max - min + 1)) );
    }
    thrust::sort(rel_a, rel_a + N, compare_int2());
    thrust::sort(rel_b, rel_b + M, compare_int2());

    // prepare device buffers
	const int threadPerBlock = 512;
	const int blocks = (N + threadPerBlock - 1) / threadPerBlock;
    int2* dev_rel_a;
    int2* dev_rel_b;
    int* out_bound;
    hipMalloc((void**) &out_bound, sizeof(int) * blocks);
    hipMalloc((void**) &dev_rel_a, sizeof(int2) * N);
    hipMalloc((void**) &dev_rel_b, sizeof(int2) * M);
	hipMemcpy(dev_rel_a, rel_a, sizeof(int2) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_rel_b, rel_b, sizeof(int2) * M, hipMemcpyHostToDevice);

	//binary_partition(rel_a, rel_b, out_bound, N, M);

}

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel( int *a, int *b, int *c ) {
 int idx = threadIdx.x + blockIdx.x * blockDim.x;
      if (idx < N) {
             int idx1 = (idx + 1) % 256;
             int idx2 = (idx + 2) % 256;
             float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
             float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
             c[idx] = (as + bs) / 2;
        }
}

void  streamTest() {

  hipDeviceProp_t  prop;
    int whichDevice;
    HANDLE_ERROR( hipGetDevice( &whichDevice ) );
    HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream0, stream1;
    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;

    // start the timers
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    // initialize the streams
    HANDLE_ERROR( hipStreamCreate( &stream0 ) );
    HANDLE_ERROR( hipStreamCreate( &stream1 ) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a0,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b0,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c0,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_a1,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b1,
                              N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c1,
                              N * sizeof(int) ) );

    // allocate host locked memory, used to stream
    HANDLE_ERROR( hipHostAlloc( (void**)&host_a,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_b,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&host_c,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault ) );

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    HANDLE_ERROR( hipEventRecord( start, 0 ) );
    // now loop over full data, in bite-sized chunks
    for (int i=0; i<FULL_DATA_SIZE; i+= N*2) {
        // enqueue copies of a in stream0 and stream1
        HANDLE_ERROR( hipMemcpyAsync( dev_a0, host_a+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream0 ) );
        HANDLE_ERROR( hipMemcpyAsync( dev_a1, host_a+i+N,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream1 ) );
        // enqueue copies of b in stream0 and stream1
        HANDLE_ERROR( hipMemcpyAsync( dev_b0, host_b+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream0 ) );
        HANDLE_ERROR( hipMemcpyAsync( dev_b1, host_b+i+N,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream1 ) );

        // enqueue kernels in stream0 and stream1
        kernel<<<N/256,256,0,stream0>>>( dev_a0, dev_b0, dev_c0 );
        kernel<<<N/256,256,0,stream1>>>( dev_a1, dev_b1, dev_c1 );

        // enqueue copies of c from device to locked memory
        HANDLE_ERROR( hipMemcpyAsync( host_c+i, dev_c0,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream0 ) );
        HANDLE_ERROR( hipMemcpyAsync( host_c+i+N, dev_c1,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream1 ) );
    }
    HANDLE_ERROR( hipStreamSynchronize( stream0 ) );
    HANDLE_ERROR( hipStreamSynchronize( stream1 ) );

    HANDLE_ERROR( hipEventRecord( stop, 0 ) );

    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );

    // cleanup the streams and memory
    HANDLE_ERROR( hipHostFree( host_a ) );
    HANDLE_ERROR( hipHostFree( host_b ) );
    HANDLE_ERROR( hipHostFree( host_c ) );
    HANDLE_ERROR( hipFree( dev_a0 ) );
    HANDLE_ERROR( hipFree( dev_b0 ) );
    HANDLE_ERROR( hipFree( dev_c0 ) );
    HANDLE_ERROR( hipFree( dev_a1 ) );
    HANDLE_ERROR( hipFree( dev_b1 ) );
    HANDLE_ERROR( hipFree( dev_c1 ) );
    HANDLE_ERROR( hipStreamDestroy( stream0 ) );
    HANDLE_ERROR( hipStreamDestroy( stream1 ) );
}

void sequentialTest() {
    hipEvent_t     start, stop;
    float           elapsedTime;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    int* host_a = new int[FULL_DATA_SIZE];
    int* host_b = new int[FULL_DATA_SIZE];
    int* host_c = new int[FULL_DATA_SIZE];
    int *dev_a, *dev_b, *dev_c;
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, FULL_DATA_SIZE * sizeof(int));
    hipMalloc( (void**)&dev_b, FULL_DATA_SIZE * sizeof(int));
    hipMalloc( (void**)&dev_c, FULL_DATA_SIZE * sizeof(int));
    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }
    HANDLE_ERROR( hipEventRecord( start, 0 ) );
	hipMemcpy(dev_a, host_a, sizeof(int) * FULL_DATA_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, sizeof(int) * FULL_DATA_SIZE, hipMemcpyHostToDevice);
    kernel<<<FULL_DATA_SIZE/256,256,0>>>( dev_a, dev_b, dev_c);
    hipMemcpyAsync( host_c, dev_c, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );
}

void
printCudaInfo() {
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
}
